
#include <hip/hip_runtime.h>
#define blocksize 256
#define gridsize 32


__global__ void mul(const float* A,const float * B, float * C, int n)
{
	int loop = n/(blocksize*gridsize);
	for(int i =0;i<loop;i++){
		*(C + i*(blocksize*gridsize) + (256 *blockIdx.x + threadIdx.x)) =  \
		*(C + i*(blocksize*gridsize) + (256 *blockIdx.x + threadIdx.x)) *  \
		*(C + i*(blocksize*gridsize) + (256 *blockIdx.x + threadIdx.x));
	} 
	if((256 *blockIdx.x + threadIdx.x)<n%(blocksize*gridsize)){
		*(C + loop*(blocksize*gridsize) + (256 *blockIdx.x + threadIdx.x)) =  \
		*(C + loop*(blocksize*gridsize) + (256 *blockIdx.x + threadIdx.x)) *  \
		*(C + loop*(blocksize*gridsize) + (256 *blockIdx.x + threadIdx.x));		
	}	
}


void eltwise_mul(const float* A,const float * B, float * C, int n)
{
	mul<<<gridsize, blocksize>>>(A, B, C, n);
}
